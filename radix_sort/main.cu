#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <ctime>
#include <math.h>

#include "sort.h"
#include "utils.h"

void cpu_sort(float* h_out, float* h_in, size_t len)
{
    for (unsigned int i = 0; i < len; ++i)
    {
        h_out[i] = h_in[i];
    }

    std::sort(h_out, h_out + len);
}

int main()
{
    srand(1);

    unsigned int N = 10000;

    for (int i = 0; i < 5; i++) {
        std::cout << "N = " << N << " ----------" << std::endl;

        // float h_in[N] = { 0.08680988, -0.44326124, -0.15096481,  0.68955225, -0.99056226, -0.75686175,
        //     0.34149817,  0.6517055,  -0.72658682,  0.15018666,  0.78264391, -0.58159578,
        //    -0.62934357, -0.78324622, -0.56060499,  0.95724756,  0.6233663 };
        // float h_in[N] = {
        //     8.68098810e-02,  -4.43261236e-01,  -1.50964811e-01,   6.89552248e-01,
        //     -9.90562260e-01,  -7.56861746e-01,   3.41498166e-01,   6.51705503e-01,
        //     -7.26586819e-01,   1.50186658e-01,   7.82643914e-01,  -5.81595778e-01,
        //     -6.29343569e-01,  -7.83246219e-01,  -5.60604990e-01,   9.57247555e-01,
        //      6.23366296e-01,  -6.56117976e-01,   6.32449508e-01,  -4.51852500e-01,
        //     -1.36591628e-01,   8.80059659e-01,   6.35298729e-01,  -3.27776104e-01,
        //     -6.49179101e-01,  -2.54335910e-01,  -9.88622963e-01,  -4.95147288e-01,
        //      5.91325045e-01,  -9.69490051e-01,   1.97686747e-01,   2.07609072e-01,
        //     -7.89704621e-01,  -2.36113116e-01,  -9.27047908e-01,   7.80823112e-01,
        //      9.61841702e-01,  -8.80116045e-01,   7.81091869e-01,   1.53803006e-01,
        //      4.84959364e-01,   2.60367870e-01,   1.63684383e-01,  -9.59121764e-01,
        //     -5.79946816e-01,   8.93697590e-02,   5.38230360e-01,  -4.98609543e-01,
        //     -4.28208619e-01,   7.04790175e-01,   9.50012982e-01,   7.69706607e-01,
        //     -2.80984312e-01,   1.97717890e-01,  -2.90408790e-01,  -3.19619566e-01,
        //     -6.43838048e-01,  -5.24611592e-01,  -9.10275459e-01,   1.08628590e-02,
        //     -2.47495085e-01,   1.85610801e-01,   2.59883761e-01,  -7.14799345e-01,
        //      8.67682576e-01,   8.92759740e-01,   2.04593316e-01,  -2.24467441e-01,
        //     -2.73624003e-01,  -5.91309428e-01,  -4.46469873e-01,  -5.06928265e-01,
        //     -6.52783990e-01,   9.33219373e-01,   9.14025187e-01,   1.95947364e-01,
        //      4.62601513e-01,  -3.19229543e-01,  -8.15888822e-01,  -7.30039626e-02,
        //      1.73977856e-02,  -8.23079646e-01,   5.60704470e-02,   9.84316051e-01,
        //     -2.09928140e-01,  -3.28807116e-01,   6.10901058e-01,   5.08697987e-01,
        //     -3.73867124e-01,   2.68073380e-01,   8.08091536e-02,  -4.06412512e-01,
        //     -7.78424203e-01,  -3.74719411e-01,  -8.60417411e-02,   3.17880154e-01,
        //     -4.91484970e-01,   2.82202512e-01,  -5.99752784e-01,   3.15249622e-01,
        //      5.56578457e-01,   5.59196770e-01,   2.20656306e-01,  -3.81999314e-01,
        //      3.95469815e-01,   7.19236612e-01,   2.50647515e-01,   9.64815676e-01,
        //      9.53000247e-01,  -6.66611731e-01,  -9.53643739e-01,  -6.78510904e-01,
        //      8.46993625e-01,   9.07099724e-01,  -5.78043163e-01,  -2.78949499e-01,
        //      9.87505242e-02,  -4.56338316e-01,  -7.87967592e-02,   3.92323136e-01,
        //      7.11793371e-04,   4.32141989e-01,   5.19118719e-02,  -9.97201979e-01,
        //     -2.10599422e-01,  -1.56660601e-02,  -1.94239333e-01,  -2.91403413e-01,
        //      1.22863892e-03
        // };
        float* h_in = new float[N];
        float* h_out = new float[N];
        unsigned int* h_idx_out = new unsigned int[N];

        for (unsigned int i = 0; i < N; i++) {
            h_in[i] = ((float) rand() / RAND_MAX - 0.5f) * 2;
            // std::cout << i << ": " << h_in[i] << std::endl;
        }

        std::clock_t start;
        start = std::clock();

        float* d_in;
        float* d_out;
        unsigned int* d_idx_out = NULL;
        checkCudaErrors(hipMalloc(&d_in, sizeof(float) * N));
        checkCudaErrors(hipMalloc(&d_out, sizeof(float) * N));
        checkCudaErrors(hipMalloc(&d_idx_out, sizeof(unsigned int) * N));
        checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(float) * N, hipMemcpyHostToDevice));

        radix_sort(d_out, d_idx_out, d_in, N);
        checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_idx_out, d_idx_out, sizeof(unsigned int) * N, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_out));
        checkCudaErrors(hipFree(d_idx_out));
        checkCudaErrors(hipFree(d_in));

        // for (unsigned int i = 0; i < N; i++) {
        //     std::cout << h_idx_out[i] << ": " << h_out[i] << std::endl;
        // }

        double gpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
        std::cout << "GPU time: " << gpu_duration << " s" << std::endl;

        float* h_out_cpu = new float[N];
        start = std::clock();
        cpu_sort(h_out_cpu, h_in, N);
        double cpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
        std::cout << "CPU time: " << cpu_duration << " s" << std::endl;

        bool match = true;
        for (unsigned int i = 0; i < N; ++i)
        {
            if (h_out_cpu[i] != h_out[i])
            {
                match = false;
                break;
            }
        }
        std::cout << "Match: " << match << std::endl;

        N *= 10;
    }
}
